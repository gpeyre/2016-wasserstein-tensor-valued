
#include <hip/hip_runtime.h>
/*
 * (c) 2013-2015 Miika Aittala, Jaakko Lehtinen, Tim Weyrich, Aalto 
 * University, University College London. This code is released under the 
 * Creative Commons Attribution-NonCommercial-ShareAlike 4.0 International 
 * license (http://creativecommons.org/licenses/by-nc-sa/4.0/).
 */

// nvcc --use_fast_math -O3 -ptx -arch=sm_30 cuda_feature.cu

__global__ void cuda_feature(unsigned int *perm, unsigned int *mins_idx, float *mins, float *fdists, unsigned int *A_feat, unsigned int *B_feat, float *A_col, float *B_col, int M2, int N, float w_col, int t)
{
/*
    int bIdx = gridDim.x*gridDim.y*blockIdx.z + gridDim.x*blockIdx.y + blockIdx.x;
    int idx = bIdx * blockDim.x * blockDim.y * blockDim.z 
            + blockDim.x * threadIdx.y
            + threadIdx.x;
*/

    //int idx = blockDim.x*gridDim.x*blockIdx.y + blockIdx.x * blockDim.x + threadIdx.x;
    int idx = blockDim.x * blockIdx.x + threadIdx.x;


    //fdists[idx] = 0;

    float dist = 0;

    unsigned int hamming = 0;
    for (int n = 0; n < N; n++)
	    hamming += __popc(A_feat[t + n*M2] ^ B_feat[idx + n*M2]);
    
    dist = (float)hamming / (float)(N*32);

    for (int n = 0; n < 3; n++)
	    dist += w_col*abs(A_col[t + n*M2] - B_col[idx + n*M2]);

    fdists[idx] = dist;

    __syncthreads();
    if (threadIdx.x != 0)
	    return;

    // Ugly recycling!
    dist = 999999;  // "infinity", eh.
    hamming = 0;

    for (int i = blockDim.x * blockIdx.x;
	 i < blockDim.x * (blockIdx.x+1); 
	 i++)
    {
	if (fdists[i] < dist)
	{
		dist = fdists[i];
		hamming = i;
	}
    }

    mins[blockIdx.x] = dist;
    mins_idx[blockIdx.x] = hamming;

    int prev = atomicInc(&mins_idx[gridDim.x], gridDim.x-1);
    if (idx != 0)
	    return;

    // Spin lock: wait until all blocks have incremented the
    // counter and it has wrapped over.
    volatile unsigned int *spin = &mins_idx[gridDim.x];
    while(*spin);

    dist = 999999;  // "infinity", eh.
    hamming = 0;
    for (int i = 0; i < gridDim.x; i++)
    {
	if (mins[i] < dist)
	{
		dist = mins[i];
		hamming = mins_idx[i] + 1; // Note: +1 for matlab
	}
    }

    perm[t] = hamming;

}

